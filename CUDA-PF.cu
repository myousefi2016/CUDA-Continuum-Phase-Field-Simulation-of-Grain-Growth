#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <vtkVersion.h>
#include <vtkSmartPointer.h>
#include <vtkXMLImageDataWriter.h>
#include <vtkImageData.h>
#include <vtkPointData.h>
#include <vtkDoubleArray.h>

#include "hip/hip_runtime.h"
#include ""

#include "utils.h"
#include "dev_matrix.h"

#define I3D(Nx,Ny,Nz,i,j,k,n) ((i)+(Nx)*(j)+(Nx)*(Ny)*(k)+(Nx)*(Ny)*(Nz)*(n))

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16
#define BLOCK_SIZE_Z 16

__global__ void pf3d_gpu(int Nx, int Ny, int Nz, int Ng, double L, double alpha, double beta, double gamma, double kappa, double ht, double hx, double hy, double hz, double *in, double *out, double *pf) 
{
	int i, j, k, n, m, P, Pm, W, E, S, N, U, B, temp;
	double lxx, lyy, lzz, sum;
        sum=0.0;
	// find i and j indices of this thread
	i = blockIdx.x*(BLOCK_SIZE_X) + threadIdx.x;
	j = blockIdx.y*(BLOCK_SIZE_Y) + threadIdx.y;
        k = blockIdx.z*(BLOCK_SIZE_Z) + threadIdx.z;

	// find indices into linear memory  
        for (n=0; n < Ng; n++) {
        P = I3D(Nx, Ny, Nz, i, j, k, n);
	W = I3D(Nx, Ny, Nz, i-1, j, k, n); E = I3D(Nx, Ny, Nz, i+1, j, k, n);
	S = I3D(Nx, Ny, Nz, i, j-1, k, n); N = I3D(Nx, Ny, Nz, i, j+1, k, n);
        B = I3D(Nx, Ny, Nz, i, j, k-1, n); U = I3D(Nx, Ny, Nz, i, j, k+1, n);
	// check that thread is within domain (not on boundary or outside domain)
	if (i > 0 && i < Nx-1 && j > 0 && j < Ny-1 && k>0 && k<Nz-1) {
		lxx = (in[W] - 2.0*in[P] + in[E])/pow(hx,2);
		lyy = (in[N] - 2.0*in[P] + in[S])/pow(hy,2);
                lzz = (in[U] - 2.0*in[P] + in[B])/pow(hz,2);

              for (m=0; m < Ng && m!=n; m++) {
                Pm = I3D(Nx, Ny, Nz, i, j, k, m);
                sum=sum+pow(in[Pm],2);
              }

        out[P] = in[P]+ht*(alpha*L*in[P]-beta*L*pow(in[P],3)-2*gamma*L*in[P]*sum+kappa*L*(lxx+lyy+lzz));
	}
        if (i==0) {
        out[P] = in[E];
        }
        if (j==0) {
        out[P] = in[N];
        }
        if (k==0) {
        out[P] = in[U];
        }
        if (i==Nx-1) {
        temp = I3D(Nx, Ny, Nz, 0, j, k, n);
        out[P] = in[temp];
        }
        if (j==Ny-1) {
        temp = I3D(Nx, Ny, Nz, i, 0, k, n);
        out[P] = in[temp];
        }
        if (k==Nz-1) {
        temp = I3D(Nx, Ny, Nz, i, j, 0, n);
        out[P] = in[temp];
        }
        pf[P] = pf[P] + pow(out[P],2);
        }
}

int main()
{
	int Nx, Ny, Nz, Nt, Ng;
        double *uh_old, *uh_new, *pfh, *tmp_h;
        int iter;
        double L, kappa, alpha, beta, gamma;
        double hx; double hy; double hz; double ht;
	dim3 numBlocks, threadsPerBlock;

	Nx = 32;
	Ny = 32;
        Nz = 32;
        Nt = 10000;
        Ng = 5;
        hx = 2.0;
        hy = 2.0;
        hz = 2.0;
        ht = 0.25;
        L = 1.0;
        kappa = 2.0;
        alpha = 1.0;
        beta = 1.0;
        gamma = 1.0;
	uh_old = dvector(Nx*Ny*Nz*Ng); uh_new = dvector(Nx*Ny*Nz*Ng); pfh = dvector(Nx*Ny*Nz);

	zero_matrix(uh_old, Nx, Ny, Nz, Ng);
	zero_matrix(uh_new, Nx, Ny, Nz, Ng);
        zero_matrix(pfh, Nx, Ny, Nz, 1);

	// initial
	initialize(uh_old, Nx, Ny, Nz, Ng);

        dev_matrix<double> ud_old(Nx, Ny, Nz, Ng); ud_old.set(uh_old, Nx, Ny, Nz, Ng);
	dev_matrix<double> ud_new(Nx, Ny, Nz, Ng); ud_new.set(uh_new, Nx, Ny, Nz, Ng);
        dev_matrix<double> tmp_d(Nx, Ny, Nz, Ng);
        dev_matrix<double> pfd(Nx, Ny, Nz, 1);
        
	numBlocks = dim3(iDivUp(Nx,BLOCK_SIZE_X), iDivUp(Ny,BLOCK_SIZE_Y), iDivUp(Nz,BLOCK_SIZE_Z));
	threadsPerBlock = dim3(BLOCK_SIZE_X, BLOCK_SIZE_Y, BLOCK_SIZE_Z);

        vtkSmartPointer<vtkImageData> imageData =
          vtkSmartPointer<vtkImageData>::New();

        imageData->SetDimensions(Nx, Ny, Nz);

        vtkSmartPointer<vtkDoubleArray> phase =
          vtkSmartPointer<vtkDoubleArray>::New();

        phase->SetNumberOfComponents(1);
        phase->SetNumberOfTuples(Nx * Ny * Nz);

	for (iter = 0; iter < Nt; iter++) {
		pf3d_gpu<<<numBlocks, threadsPerBlock>>>(Nx, Ny, Nz, Ng, L, alpha, beta, gamma, kappa, ht, hx, hy, hz, ud_old.getData(), ud_new.getData(), pfd.getData());
        tmp_d = ud_new;
        ud_new = ud_old;
        ud_old = tmp_d;
        tmp_h = uh_new;
        uh_new = uh_old;
        uh_old = tmp_h;
        char myfile[16];
        sprintf(myfile, "myfile_%d.vti", iter);
        pfd.get(pfh, Nx, Ny, Nz, 1);
        for (i=0; i < Nx; i++) {
        for (j=0; j < Ny; j++) {
        for (k=0; k < Nz; k++) {
        P = I3D(Nx, Ny, Nz, i, j, k, 0);
        phase->SetValue(P, pfh[P]);
        }
        }
        }
        imageData->GetPointData()->AddArray(phase);
        phase->SetName("Phase Field");
        vtkSmartPointer<vtkXMLImageDataWriter> writer =
          vtkSmartPointer<vtkXMLImageDataWriter>::New();

        writer->SetFileName(myfile);
    #if VTK_MAJOR_VERSION <= 5
        writer->SetInputConnection(imageData->GetProducerPort());
    #else
        writer->SetInputData(imageData);
    #endif
        writer->Write();
	} 
	hipDeviceSynchronize();
}


